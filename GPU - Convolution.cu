
/**
*   CS6023: GPU Programming 
*   Assignment 2
*   
*   Please don't change any existing code in this file.
*
*   Please add necessary memory APIs for your implementation. Use cudaFree() 
*   to free up memory as soon as you're done with an allocation. 
*   This will ensure that you don't run out of memory while running
*   large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;

typedef long long ll;

__global__ void dKernel(long int *h_mat,long int *h_filter, long int *h_ans,int k){
   
    int mid = k/2;
    int rowUp = blockIdx.x - mid;
    int rowDown = blockIdx.x + mid;
    int colLeft = threadIdx.x - mid;
    int colRight = threadIdx.x + mid;
    long int sum=0;
    int f1=0;
    int f2=0;
    if(rowUp < 0){
      rowUp = 0;
      f1 = mid - blockIdx.x;
    }
    if(rowDown >= gridDim.x){
      rowDown = gridDim.x-1;
    }
    if(colLeft < 0){
      colLeft =0;
      f2 = mid - threadIdx.x;
    }
   if(colRight >= blockDim.x){
    colRight = blockDim.x-1;
   }
    for(int i = rowUp,k1=f1 ; i<=rowDown ; i++,k1++){
      for(int j= colLeft,k2=f2; j<=colRight ; j++,k2++){
        {
          sum += h_mat[i*blockDim.x+j] * h_filter[k1* k + k2 ];
        }
      }
    }
    h_ans[blockIdx.x * blockDim.x+threadIdx.x] = sum;
}
int main(int argc, char** argv) {

    int m,n,k;
    cin>>m>>n>>k; 
    long int* h_mat = new long int[m * n];
    long int* h_filter = new long int[k * k];
    long int* h_ans = new long int[m * n];

    for (long int i = 0; i < m * n; i++) {
        cin>>h_mat[i];
    }

    for (long int i = 0; i < k * k; i++) {
        cin>>h_filter[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    **/

    /****************************************************Start Here***********************************************************/
    
    long int * gpuH_mat;
    long int * gpuH_filter;
    long int * gpuH_ans;

    hipMalloc(&gpuH_mat,m*n*sizeof(long int));
    hipMalloc(&gpuH_filter,k*k*sizeof(long int));
    hipMalloc(&gpuH_ans,m*n*sizeof(long int));

    hipMemcpy(gpuH_mat,h_mat,m*n*sizeof(long int),hipMemcpyHostToDevice);
    hipMemcpy(gpuH_filter,h_filter,k*k*sizeof(long int),hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();//keep it just before the kernel launch

    dKernel<<<m,n>>>(gpuH_mat,gpuH_filter,gpuH_ans,k);
    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();//keep it just after the kernel launch

    hipMemcpy(h_ans,gpuH_ans,m*n*sizeof(long int),hipMemcpyDeviceToHost);
      
    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    std::chrono::duration<double> elapsed1 = end - start;
    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */


    
    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < m; i++) {
            for (long int j = 0; j < n; j++) {
                file << h_ans[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}